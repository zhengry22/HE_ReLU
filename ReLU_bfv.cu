#include "hip/hip_runtime.h"
#pragma once
#include "ReLU_bfv.h"
#include "../Polynomial_Calc/polynomial.h"
#include "../Polynomial_Calc/SiLU.h"
#include <vector>
#include <cmath>
using namespace troy;
using namespace std;
#define MYDEBUG

extern const size_t poly_modulus_degree;
// extern const vector<Modulus> coeff_modulus;
extern const size_t plain_modulus;


void poly_relu(const auto &encoder, const Evaluator &evaluator, const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    /*
        This is the initial version of the poly_relu function.
        In the first version, we use x^2 + 7x + 10 to simulate the pt's polynomial * 15.
        In order to decrypt, we should first obtain the ciphertext and then divide the result by 15.
    */

    // First encode x + 2
    Ciphertext x_plus_two = x_encrypted; 
    Plaintext plain_two;
    encoder.encode_polynomial({2}, plain_two);
    evaluator.add_plain_inplace(x_plus_two, plain_two);

    // // Next encode x + 5
    Ciphertext x_plus_five = x_encrypted;
    Plaintext plain_five;
    encoder.encode_polynomial({5}, plain_five);
    evaluator.add_plain_inplace(x_plus_five, plain_five);

    // At last, multiply the two polynomials together

    evaluator.multiply(x_plus_two, x_plus_five, encrypted_result);
} 

void test_4degree_horner(const auto &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    
    /*
        This function is used to test whether using 4 degree polynomial has a problem
    */
    cout << "Testing horner's method using a 4 degree polynomial: " << endl;
    Plaintext zero;
    Ciphertext my_cipher = x_encrypted; // x
    encoder.encode_polynomial({0}, zero);

    evaluator.multiply_plain_inplace(my_cipher, zero); // x * 0
    evaluator.relinearize_inplace(my_cipher, relin_keys);
    Plaintext this_pt;
    encoder.encode_polynomial({1}, this_pt);
    evaluator.add_plain_inplace(my_cipher, this_pt); // x * 0 + 1

    Ciphertext mid;
    evaluator.multiply(my_cipher, x_encrypted, mid); // 1 * x
    evaluator.relinearize_inplace(mid, relin_keys); 
    my_cipher = mid;

    encoder.encode_polynomial({0}, this_pt); // 1 * x + 0
    evaluator.add_plain_inplace(my_cipher, this_pt);

    evaluator.multiply(my_cipher, x_encrypted, mid); // x^2
    evaluator.relinearize_inplace(mid, relin_keys); 
    my_cipher = mid;

    evaluator.multiply(my_cipher, x_encrypted, mid); // x^2
    evaluator.relinearize_inplace(mid, relin_keys); 
    my_cipher = mid;

    //     evaluator.multiply(my_cipher, x_encrypted, mid); // x^2
    // evaluator.relinearize_inplace(mid, relin_keys); 
    // my_cipher = mid;
    // encoder.encode_polynomial({8181}, this_pt); // x*2 - 11
    // evaluator.add_plain_inplace(my_cipher, this_pt);

    // evaluator.multiply(my_cipher, x_encrypted, mid); // x^3 - 11x
    // evaluator.relinearize_inplace(mid, relin_keys); 
    // my_cipher = mid;

    // encoder.encode_polynomial({8168}, this_pt); // x^3 - 11x - 24
    // evaluator.add_plain_inplace(my_cipher, this_pt);

    // evaluator.multiply(my_cipher, x_encrypted, mid); // x^4 - 11x^2 - 24x
    // evaluator.relinearize_inplace(mid, relin_keys); 
    // my_cipher = mid;

    // encoder.encode_polynomial({0}, this_pt); // x^4 - 11x^2 - 24x
    // evaluator.add_plain_inplace(my_cipher, this_pt);

    encrypted_result = my_cipher;
}

void horner(const auto &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const EncryptPolynomial &encpoly, const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    /*
        This function has bug in it! or the way we decrypt!
    */
    
    cout << "Using horners' method to calculate: " << endl;
    
    /*
        In order to calculate the final result, we may use the linear horner's algorithm. 
        Note that this is not likely the most efficient way, but is convenient for testing
    */

    cout << "Generate relinearization keys." << endl;

    int deg = encpoly.poly.get_degree();
    Plaintext zero;
    Ciphertext my_cipher = x_encrypted;
    encoder.encode_polynomial({0}, zero);
    for (int i = deg; i >= 0; i--) {
        // First multiple x and then add coeff[i]
        if (i == deg) {
            evaluator.multiply_plain_inplace(my_cipher, zero);
            evaluator.relinearize_inplace(my_cipher, relin_keys);
        }   
        else {
            Ciphertext mid;
            evaluator.multiply(my_cipher, x_encrypted, mid);
            evaluator.relinearize_inplace(mid, relin_keys);
            my_cipher = mid;
        }
        uint64_t this_coeff = encpoly.poly.get_coeff_by_rank(i);
        Plaintext this_pt;
        encoder.encode_polynomial({this_coeff}, this_pt);
        evaluator.add_plain_inplace(my_cipher, this_pt);
    }

    encrypted_result = my_cipher;
}


int main() {
    int deg;
    cout << "Input deg: " << endl;
    cin >> deg;

    // Generate the polynomial for approximation
    Taylor<double, double> taylor(deg, silu);
    Polynomial<double> poly = taylor.generate_approx(deg, 0);
    poly.prune();
    poly.check();
    EncryptPolynomial encpoly = round_polynomial(poly);
    encpoly.show();


    // These are the parameters used for encryption: ReLU(x_) and ReLU(y_)
    uint64_t x_ = 3;

    // Set the encryption scheme to be BFV
    EncryptionParameters parms(SchemeType::BFV);

    // Set the modulus of the polynomial modulus: x^d + 1, where d is a power of 2
    parms.set_poly_modulus_degree(poly_modulus_degree);

    // Set the coeff_modulus
    parms.set_coeff_modulus(CoeffModulus::bfv_default(poly_modulus_degree, SecurityLevel::Classical128));

    // Set the plain_modulus
    parms.set_plain_modulus(plain_modulus);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    auto encoder = BatchEncoder(context);
    context->to_device_inplace();
    encoder.to_device_inplace();

    KeyGenerator keygen(context);
    SecretKey secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);

    Encryptor encryptor(context);
    encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);

    /*
        First, encode x to poly_relu as poly_relu(x), 
        where poly_relu is the polynomial we use to simulate ReLU.
        This procedure should be done via these steps: 
            1. Encode the number x to 'x as a polynomial'
    */
    Plaintext x_plain;
    encoder.encode_polynomial({x_}, x_plain);
    /*
            2. Encrypt x
    */
    Ciphertext x_encrypted;
    encryptor.encrypt_asymmetric(x_plain, x_encrypted);
    /*
            3. Now we get the ciphertext c, calculate d, so that Relu(Dec(c)) = Dec(d)
    */
    Ciphertext encrypted_result;

    // poly_relu(encoder, evaluator, x_encrypted, encrypted_result);

    RelinKeys relin_keys = keygen.create_relin_keys(false);
    //horner(encoder, evaluator, relin_keys, encpoly, x_encrypted, encrypted_result);
    test_4degree_horner(encoder, evaluator, relin_keys, x_encrypted, encrypted_result);
    /*
        Decrypt `encrypted result` and divide the result by 15
    */

    Plaintext decrypted_result;
    decryptor.decrypt(encrypted_result, decrypted_result);
    std::string my_answer = decrypted_result.to_string();
    int int_value = std::stoi(my_answer, nullptr, 16);

    //int_value = ((plain_modulus >> 1) > int_value) ? int_value : int_value - plain_modulus;

    cout << int_value << " " << (double)int_value / (double)(encpoly.k) << endl;

    long long result = std::lround((double)int_value / (double)(encpoly.k));

    // 输出结果
    std::cout << "Relu(x) = " << result << endl;
    

    return 0;
}