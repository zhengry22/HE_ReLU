#include "hip/hip_runtime.h"
#pragma once
#include "ReLU_ckks.h"
#include "../examples/examples.h"
#include "../Polynomial_Calc/polynomial.h"
#include "../Polynomial_Calc/SiLU.h"
#include <iostream>
#include <chrono>
#include <ctime>
#include <codecvt>
#include <locale>
//#include "./matplotlib-cpp/matplotlibcpp.h"
//#define MYDEBUG
#define BATCH_SIZE 40
#define PARTS 2
using namespace troy;
using namespace std;
//namespace plt = matplotlibcpp;
/*
    Try to calculate the cipher with a fixed polynomial 
*/ 
void approx_with_fix(const CKKSEncoder &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const double &scale,
                    const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    
    // Prepare the plaintext for 2, 5, and 1
    Plaintext plain_2, plain_5, plain_1;
    encoder.encode_float64_single(2.0, std::nullopt, scale, plain_2);
    encoder.encode_float64_single(5.0, std::nullopt, scale, plain_5);
    encoder.encode_float64_single(1.0, std::nullopt, scale, plain_1);

    // Compute x + 2
    Ciphertext x_plus_2 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_2, plain_2);

    // Compute x + 5
    Ciphertext x_plus_5 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_5, plain_5);

    // Compute x + 1
    Ciphertext x_plus_1 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_1, plain_1);

    // Compute (x + 2) * (x + 5)
    evaluator.multiply_inplace(x_plus_2, x_plus_5);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Compute (x + 2) * (x + 5) * (x + 1)
    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Compute (x + 2) * (x + 5) * (x + 1) * (x + 1)
    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Store the result
    encrypted_result = x_plus_2;
}


void horner(const CKKSEncoder &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const double &scale,
            const Polynomial<double> &poly, const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {

    /*
        Conduct the horner's algorithm in linear complexity
    */
    Ciphertext my_cipher = x_encrypted; // set the initial value to be x
    size_t poly_deg = poly.get_degree();
    Plaintext plain_coeff;
    double largest_coeff = poly.get_coeff_by_rank(poly_deg);
    //double largest_coeff = 1.0;
    encoder.encode_float64_single(largest_coeff, std::nullopt, scale, plain_coeff);
    evaluator.multiply_plain_inplace(my_cipher, plain_coeff);
    evaluator.rescale_to_next_inplace(my_cipher); // a_n*x

    for (int i = poly_deg - 1; i >= 0; i--) {
        if (i != poly_deg - 1) {
            // multiply x
            Ciphertext this_x = x_encrypted;
            // First change scale 
            evaluator.mod_switch_to_inplace(this_x, my_cipher.parms_id());
            this_x.scale() = my_cipher.scale();
            // Second multiply
            evaluator.multiply_inplace(my_cipher, this_x);
            evaluator.relinearize_inplace(my_cipher, relin_keys);
            evaluator.rescale_to_next_inplace(my_cipher);
        }
        // Add const
        Plaintext this_coeff;
        encoder.encode_float64_single(poly.get_coeff_by_rank((size_t)(i)), std::nullopt, my_cipher.scale(), this_coeff);
        //encoder.encode_float64_single(0.0, std::nullopt, my_cipher.scale(), this_coeff);
        evaluator.add_plain_inplace(my_cipher, this_coeff);
    }

    encrypted_result = my_cipher;
}


void multiply_x(const CKKSEncoder &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, 
    const double &scale, const int &degree, const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    Ciphertext my_cipher = x_encrypted; // set the initial value to be x

    int deg_left = degree - 1; // Its x ^ 1 at the beginning
    while (deg_left > 0) {
        if (deg_left % 2 == 1) {
            // multiply x
            Ciphertext this_x = x_encrypted;
            // First change scale 
            evaluator.mod_switch_to_inplace(this_x, my_cipher.parms_id());
            this_x.scale() = my_cipher.scale();
            // Second multiply
            evaluator.multiply_inplace(my_cipher, this_x);
            evaluator.relinearize_inplace(my_cipher, relin_keys);
            evaluator.rescale_to_next_inplace(my_cipher);
            deg_left -= 1;
        }
        else {
            Ciphertext tmp;
            evaluator.square(my_cipher, tmp);
            evaluator.relinearize_inplace(tmp, relin_keys);
            evaluator.rescale_to_next_inplace(tmp);
            my_cipher = tmp;
            deg_left >>= 1;  
        }    
    }

    encrypted_result = my_cipher;
}


int main() {

    int deg;
    cout << "Input deg: " << endl;
    cin >> deg;

    vector<size_t> mod_chain;
    for (int i = 0; i < deg + 2; i++) {
        if (i == 0) {
            mod_chain.push_back(60);
        }
        else {
            mod_chain.push_back(40);
        }
    }

    EncryptionParameters parms(SchemeType::CKKS);

    Remez<double, double> my_p(deg, gelu_and_sqplus);
    Polynomial<double> poly = my_p.generate_approx(deg, 0);
    poly.prune();
    poly.check();

    size_t poly_modulus_degree = 32768;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::create(poly_modulus_degree, mod_chain));
    // Change this modulus chain if we need higher degree


    double scale = pow(2.0, 40);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();
    //cout << "Number of slots: " << slot_count << endl;

    context->to_device_inplace();
    encoder.to_device_inplace();

    KeyGenerator keygen(context);
    auto secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);
    RelinKeys relin_keys = keygen.create_relin_keys(false);
    GaloisKeys gal_keys = keygen.create_galois_keys(false);
    Encryptor encryptor(context); encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);


    vector<complex<double>> input;
    input.reserve(slot_count);
    double curr_point = -5;
    //double step_size = 1.0 / (static_cast<double>(slot_count) - 1);
    double step_size = (double)10 / (double)BATCH_SIZE;
    for (size_t i = 0; i < BATCH_SIZE; i++)
    {
        input.push_back(curr_point);
        curr_point += step_size;
    }
    cout << "Input vector: " << endl;
    print_vector(input, BATCH_SIZE, 7);

    /*
        Try to calculate th
    */


    Plaintext x_plain;
    cout << "Encode input vectors." << endl;
    encoder.encode_complex64_simd(input, std::nullopt, scale, x_plain);
    Ciphertext x_encrypted;
    encryptor.encrypt_asymmetric(x_plain, x_encrypted);
    Ciphertext encrypted_result;

    /*
        Create multiple threads
    */
    vector<Polynomial<double>> fragments;
    for (int i = 0; i < PARTS; i++) {
        Polynomial<double> tmp = poly.slice((poly.get_degree() + 1) * i / PARTS, ((poly.get_degree() + 1) * (i + 1) / PARTS) - 1);
        fragments.push_back(tmp);
        tmp.check();
    }

    auto start = std::chrono::high_resolution_clock::now();
    vector<Ciphertext> ciphers(2 * PARTS - 1);

    vector<thread> threads;
    mutex mtx;

    for (int i = 0; i < (2 * PARTS - 1); i++) {
        if (i % 2 == 0) {
            threads.push_back(thread(horner, cref(encoder), cref(evaluator), cref(relin_keys), 
                                    cref(scale), cref(fragments[i / 2]), cref(x_encrypted), ref(ciphers[i])));
        } else {
            int degree = poly.get_degree() + 1;
            int part_degree = degree * (i + 1) / (PARTS * 2);
            threads.push_back(thread(multiply_x, cref(encoder), cref(evaluator), cref(relin_keys), 
                                    cref(scale), cref(part_degree), cref(x_encrypted), ref(ciphers[i])));
        }
    }

    for (auto &t : threads) {
        if (t.joinable()) {
            t.join();
        }
    }

#ifdef MYDEBUG
    cout << "Debugging..." << endl;
    for (int i = 0; i < (2 * PARTS - 1); i++) {
        cout << "i = " << i << endl;
        Plaintext plain_result;
        decryptor.decrypt(ciphers[i], plain_result);
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
        vector<complex<double>> result;
        encoder.decode_complex64_simd(plain_result, result);
        for (int j = 0; j < BATCH_SIZE; j++) {
            //cout << "x: " << input[i].real() << "       Relu(x): " << relu(input[i].real()) << "      SiLU(x): " << silu(input[i].real()) << "       计算结果：" << result[i].real() << endl;
            cout << "x: " << input[j].real() << "       Relu(x): " << relu(input[j].real()) << "       计算结果：" << result[j].real();
            if (i % 2 == 0) {
                cout << "     True value: " << fragments[i / 2].get_poly_value(input[j].real());
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "Debugging end..." << endl;
#endif

    vector<thread> mult(PARTS - 1);

    for (int i = 1; i < (2 * PARTS - 1); i += 2) {
        // multiply x
        mult.push_back(thread([&evaluator, &relin_keys, &ciphers, i]() mutable {
            Ciphertext this_x = ciphers[i];
            // First change scale 
            evaluator.mod_switch_to_inplace(this_x, ciphers[i + 1].parms_id());
            this_x.scale() = ciphers[i + 1].scale();
            // Second multiply
            evaluator.multiply_inplace(ciphers[i + 1], this_x);
            evaluator.relinearize_inplace(ciphers[i + 1], relin_keys);
            evaluator.rescale_to_next_inplace(ciphers[i + 1]);
        }));
    }

    for (auto &t : mult) {
        if (t.joinable()) {
            t.join();
        }
    }

    Ciphertext enc;

    

    for (int i = 2; i < 2 * PARTS - 1; i += 2) {
        Ciphertext this_x = ciphers[i - 2];
        evaluator.mod_switch_to_inplace(this_x, ciphers[i].parms_id());
        this_x.scale() = ciphers[i].scale();
        evaluator.add_inplace(ciphers[i], this_x);
        if ((i + 2) > (2 * PARTS - 1)) {
            enc = ciphers[i];
        }
    }

    //approx_with_fix(encoder, evaluator, relin_keys, scale, x_encrypted, encrypted_result);
    
    // horner(encoder, evaluator, relin_keys, scale, poly, x_encrypted, encrypted_result);
    
    /*
    Decrypt, decode, and print the result.
    */
    Plaintext plain_result;
    decryptor.decrypt(enc, plain_result);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    vector<complex<double>> result;
    encoder.decode_complex64_simd(plain_result, result);
    //print_vector(result, 16, 7);

    /*
        Ploting the curve: 
    */

    vector<double> x_coord, relu_y, poly_y;
    for (int i = 0; i < BATCH_SIZE; i++) {
        //cout << "x: " << input[i].real() << "       Relu(x): " << relu(input[i].real()) << "      SiLU(x): " << silu(input[i].real()) << "       计算结果：" << result[i].real() << endl;
        cout << "x: " << input[i].real() << "       Relu(x): " << relu(input[i].real()) << "       计算结果：" << result[i].real() << endl;
        x_coord.push_back(input[i].real());
        relu_y.push_back(relu(input[i].real()));
        poly_y.push_back(result[i].real());
    }
    std::cout << "运行时间: " << (double)duration.count() / (double)1000 << " ms" << std::endl;
    // Py_Initialize();
    
    // // 获取 Python 解释器路径
    // const wchar_t* python_executable_w = Py_GetProgramName();
    
    // // 将 wchar_t* 转换为 std::string
    // std::wstring_convert<std::codecvt_utf8<wchar_t>> converter;
    // std::string python_executable = converter.to_bytes(python_executable_w);

    // std::cout << "Python executable path: " << python_executable << std::endl;
    
    // Py_Finalize();

    // plt::plot(x_coord, relu_y);
    // plt::plot(x_coord, poly_y);  
    // plt::show();
    return 0;

}
