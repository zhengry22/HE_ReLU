#include "hip/hip_runtime.h"
#pragma once
#include "ReLU_ckks.h"
#include "../examples/examples.h"
#include "../Polynomial_Calc/polynomial.h"
#include "../Polynomial_Calc/SiLU.h"
#include <iostream>

using namespace troy;
using namespace std;

double relu(double x) {
    return x > 0 ? x : 0;
}

/*
    Try to calculate the cipher with a fixed polynomial x^2 + 7x + 10
*/ 
void approx_with_fix(const CKKSEncoder &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const double &scale,
                    const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    

    // prepare the plaintext for 2 and 5
    Plaintext plain_2, plain_5;
    encoder.encode_float64_single(2.0, std::nullopt, scale, plain_2);
    encoder.encode_float64_single(5.0, std::nullopt, scale, plain_5);


    // x + 2
    Ciphertext x_plus_2 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_2, plain_2);
    //evaluator.relinearize_inplace(x_plus_2, relin_keys);
    //evaluator.rescale_to_next_inplace(x_plus_2);

    // x + 5
    Ciphertext x_plus_5 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_5, plain_5);
    //evaluator.relinearize_inplace(x_plus_5, relin_keys);
    //evaluator.rescale_to_next_inplace(x_plus_5);

    evaluator.multiply_inplace(x_plus_2, x_plus_5);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);
    encrypted_result = x_plus_2;

}

int main() {

    EncryptionParameters parms(SchemeType::CKKS);

    size_t poly_modulus_degree = 8192;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::create(poly_modulus_degree, { 60, 40, 40, 60 }));
    // Change this modulus chain if we need higher degree

    double scale = pow(2.0, 40);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();
    cout << "Number of slots: " << slot_count << endl;

    context->to_device_inplace();
    encoder.to_device_inplace();

    KeyGenerator keygen(context);
    auto secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);
    RelinKeys relin_keys = keygen.create_relin_keys(false);
    GaloisKeys gal_keys = keygen.create_galois_keys(false);
    Encryptor encryptor(context); encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);


    vector<complex<double>> input;
    input.reserve(slot_count);
    double curr_point = -2;
    //double step_size = 1.0 / (static_cast<double>(slot_count) - 1);
    double step_size = 0.25;
    for (size_t i = 0; i < 16; i++)
    {
        input.push_back(curr_point);
        curr_point += step_size;
    }
    cout << "Input vector: " << endl;
    print_vector(input, 16, 7);

    cout << "Evaluating polynomial x^2 + 7x + 10 ..." << endl;

    /*
        Try to calculate th
    */


    Plaintext x_plain;
    print_line(__LINE__);
    cout << "Encode input vectors." << endl;
    encoder.encode_complex64_simd(input, std::nullopt, scale, x_plain);
    Ciphertext x_encrypted;
    encryptor.encrypt_asymmetric(x_plain, x_encrypted);
    Ciphertext encrypted_result;
    approx_with_fix(encoder, evaluator, relin_keys, scale, x_encrypted, encrypted_result);

    /*
    Decrypt, decode, and print the result.
    */
    Plaintext plain_result;
    decryptor.decrypt(encrypted_result, plain_result);
    print_line(__LINE__);
    vector<complex<double>> result;
    encoder.decode_complex64_simd(plain_result, result);
    cout << "    + Computed result ...... Correct." << endl;
    cout << "    size of the result: " << result.size() << endl;
    //print_vector(result, 16, 7);
    for (int i = 0; i < 16; i++) {
        cout << "Relu(x) / calc: " << relu(input[i].real()) << " " << result[i].real() / 15 << endl;
    }
    cout << endl;
    return 0;

}
