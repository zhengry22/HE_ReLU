#include "hip/hip_runtime.h"
#pragma once
#include "ReLU_ckks.h"
#include "../examples/examples.h"
#include "../Polynomial_Calc/polynomial.h"
#include "../Polynomial_Calc/SiLU.h"
#include <iostream>

using namespace troy;
using namespace std;

double relu(double x) {
    return x > 0 ? x : 0;
}

/*
    Try to calculate the cipher with a fixed polynomial 
*/ 
void approx_with_fix(const CKKSEncoder &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const double &scale,
                    const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    
    // Prepare the plaintext for 2, 5, and 1
    Plaintext plain_2, plain_5, plain_1;
    encoder.encode_float64_single(2.0, std::nullopt, scale, plain_2);
    encoder.encode_float64_single(5.0, std::nullopt, scale, plain_5);
    encoder.encode_float64_single(1.0, std::nullopt, scale, plain_1);

    // Compute x + 2
    Ciphertext x_plus_2 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_2, plain_2);

    // Compute x + 5
    Ciphertext x_plus_5 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_5, plain_5);

    // Compute x + 1
    Ciphertext x_plus_1 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_1, plain_1);

    // Compute (x + 2) * (x + 5)
    evaluator.multiply_inplace(x_plus_2, x_plus_5);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Compute (x + 2) * (x + 5) * (x + 1)
    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Compute (x + 2) * (x + 5) * (x + 1) * (x + 1)
    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Store the result
    encrypted_result = x_plus_2;
}


void horner(const auto &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const double &scale,
            const Polynomial<double> poly, const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {

    /*
        Conduct the horner's algorithm in linear complexity
    */
    Ciphertext my_cipher = x_encrypted; // set the initial value to be x
    size_t poly_deg = poly.get_degree();
    Plaintext plain_coeff;
    double largest_coeff = poly.get_coeff_by_rank(poly_deg);
    //double largest_coeff = 1.0;
    cout << "First coeff is: " << largest_coeff << endl;
    encoder.encode_float64_single(largest_coeff, std::nullopt, scale, plain_coeff);
    evaluator.multiply_plain_inplace(my_cipher, plain_coeff);
    evaluator.rescale_to_next_inplace(my_cipher); // a_n*x

    for (int i = poly_deg - 1; i >= 0; i--) {
        if (i != poly_deg - 1) {
            // multiply x
            Ciphertext this_x = x_encrypted;
            // First change scale 
            evaluator.mod_switch_to_inplace(this_x, my_cipher.parms_id());
            this_x.scale() = my_cipher.scale();
            // Second multiply
            evaluator.multiply_inplace(my_cipher, this_x);
            evaluator.relinearize_inplace(my_cipher, relin_keys);
            evaluator.rescale_to_next_inplace(my_cipher);
        }
        // Add const
        Plaintext this_coeff;
        encoder.encode_float64_single(poly.get_coeff_by_rank((size_t)(i)), std::nullopt, my_cipher.scale(), this_coeff);
        //encoder.encode_float64_single(0.0, std::nullopt, my_cipher.scale(), this_coeff);
        evaluator.add_plain_inplace(my_cipher, this_coeff);
    }

    encrypted_result = my_cipher;
}


int main() {

    int deg;
    cout << "Input deg: " << endl;
    cin >> deg;

    vector<size_t> mod_chain;
    for (int i = 0; i < deg + 2; i++) {
        if (i == 0) {
            mod_chain.push_back(60);
        }
        else {
            mod_chain.push_back(40);
        }
    }

    EncryptionParameters parms(SchemeType::CKKS);

    Taylor<double, double> taylor(deg, silu);
    Polynomial<double> poly = taylor.generate_approx(deg, 0);
    poly.prune();
    poly.check();

    size_t poly_modulus_degree = 32768;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::create(poly_modulus_degree, mod_chain));
    // Change this modulus chain if we need higher degree

    cout << "Chain completed! " << endl;

    double scale = pow(2.0, 40);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();
    cout << "Number of slots: " << slot_count << endl;

    context->to_device_inplace();
    encoder.to_device_inplace();

    KeyGenerator keygen(context);
    auto secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);
    RelinKeys relin_keys = keygen.create_relin_keys(false);
    GaloisKeys gal_keys = keygen.create_galois_keys(false);
    Encryptor encryptor(context); encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);


    vector<complex<double>> input;
    input.reserve(slot_count);
    double curr_point = -2;
    //double step_size = 1.0 / (static_cast<double>(slot_count) - 1);
    double step_size = 0.25;
    for (size_t i = 0; i < 16; i++)
    {
        input.push_back(curr_point);
        curr_point += step_size;
    }
    cout << "Input vector: " << endl;
    print_vector(input, 16, 7);

    cout << "Evaluating polynomial (x + 2)(x + 5)(x + 1)(x + 1)..." << endl;

    /*
        Try to calculate th
    */


    Plaintext x_plain;
    print_line(__LINE__);
    cout << "Encode input vectors." << endl;
    encoder.encode_complex64_simd(input, std::nullopt, scale, x_plain);
    Ciphertext x_encrypted;
    encryptor.encrypt_asymmetric(x_plain, x_encrypted);
    Ciphertext encrypted_result;
    //approx_with_fix(encoder, evaluator, relin_keys, scale, x_encrypted, encrypted_result);
    cout << "In horner!" << endl;
    horner(encoder, evaluator, relin_keys, scale, poly, x_encrypted, encrypted_result);

    /*
    Decrypt, decode, and print the result.
    */
    Plaintext plain_result;
    decryptor.decrypt(encrypted_result, plain_result);
    print_line(__LINE__);
    vector<complex<double>> result;
    encoder.decode_complex64_simd(plain_result, result);
    //print_vector(result, 16, 7);
    for (int i = 0; i < 16; i++) {
        cout << "Relu(x) / calc: " << relu(input[i].real()) << " " << result[i].real() << endl;
    }
    cout << endl;
    return 0;

}
