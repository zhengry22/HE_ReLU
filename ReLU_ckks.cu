#include "hip/hip_runtime.h"
#pragma once
#include "ReLU_ckks.h"
#include "../examples/examples.h"
#include "../Polynomial_Calc/polynomial.h"
#include "../Polynomial_Calc/SiLU.h"
#include <iostream>

using namespace troy;
using namespace std;

int main() {

    EncryptionParameters parms(SchemeType::CKKS);

    size_t poly_modulus_degree = 8192;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::create(poly_modulus_degree, { 60, 40, 40, 60 }));

    double scale = pow(2.0, 40);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();
    cout << "Number of slots: " << slot_count << endl;

    context->to_device_inplace();
    encoder.to_device_inplace();

    KeyGenerator keygen(context);
    auto secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);
    RelinKeys relin_keys = keygen.create_relin_keys(false);
    GaloisKeys gal_keys = keygen.create_galois_keys(false);
    Encryptor encryptor(context); encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);


    vector<complex<double>> input;
    input.reserve(slot_count);
    double curr_point = 0;
    //double step_size = 1.0 / (static_cast<double>(slot_count) - 1);
    double step_size = 0.25;
    for (size_t i = 0; i < 10; i++)
    {
        input.push_back(curr_point);
        curr_point += step_size;
    }
    cout << "Input vector: " << endl;
    print_vector(input, 10, 7);

    cout << "Evaluating polynomial PI*x^3 + 0.4x + 1 ..." << endl;

    /*
    We create plaintexts for PI, 0.4, and 1 using an overload of CKKSEncoder::encode
    that encodes the given floating-point value to every slot in the vector.
    */
    Plaintext plain_coeff2, plain_coeff1, plain_coeff0;
    encoder.encode_float64_single(3.0, std::nullopt, scale, plain_coeff2);
    encoder.encode_float64_single(2.0, std::nullopt, scale, plain_coeff1);
    encoder.encode_float64_single(1.0, std::nullopt, scale, plain_coeff0);

    Plaintext x_plain;
    print_line(__LINE__);
    cout << "Encode input vectors." << endl;
    encoder.encode_complex64_simd(input, std::nullopt, scale, x_plain);
    Ciphertext x1_encrypted;
    encryptor.encrypt_asymmetric(x_plain, x1_encrypted);

    Ciphertext x3_encrypted;
    print_line(__LINE__);
    cout << "Compute x^2 and relinearize:" << endl;
    evaluator.square(x1_encrypted, x3_encrypted);
    evaluator.relinearize_inplace(x3_encrypted, relin_keys);
    cout << "    + Scale of x^2 before rescale: " << log2(x3_encrypted.scale()) << " bits" << endl;

    print_line(__LINE__);
    cout << "Rescale x^2." << endl;
    evaluator.rescale_to_next_inplace(x3_encrypted);
    cout << "    + Scale of x^2 after rescale: " << log2(x3_encrypted.scale()) << " bits" << endl;


    // Ensure the plain_coeff2 has the same parms_id as the rescaled x3_encrypted
    Plaintext plain_coeff2_adjusted;
    encoder.encode_float64_single(3.0, x3_encrypted.parms_id(), x3_encrypted.scale(), plain_coeff2_adjusted);

    // Then multiply the rescaled ciphertext with the adjusted plaintext
    Ciphertext x1_encrypted_coeff3;
    evaluator.multiply_plain(x3_encrypted, plain_coeff2_adjusted, x1_encrypted_coeff3);

    cout << "    + Scale of 3x^2 before rescale: " << log2(x1_encrypted_coeff3.scale()) << " bits" << endl;
    evaluator.rescale_to_next_inplace(x1_encrypted_coeff3);
    cout << "    + Scale of 3x^2 after rescale: " << log2(x1_encrypted_coeff3.scale()) << " bits" << endl;
    print_line(__LINE__);
    /*
    Decrypt, decode, and print the result.
    */
    Plaintext plain_result;
    decryptor.decrypt(x1_encrypted_coeff3, plain_result);
    print_line(__LINE__);
    vector<complex<double>> result;
    encoder.decode_complex64_simd(plain_result, result);
    cout << "    + Computed result ...... Correct." << endl;
    cout << "    size of the result: " << result.size() << endl;
    print_vector(result, 10, 7);

    return 0;

}
