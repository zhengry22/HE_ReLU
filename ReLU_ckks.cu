#include "hip/hip_runtime.h"
#include "ReLU_ckks.h"
#include "../examples/examples.h"
#include "../Polynomial_Calc/polynomial.h"
#include "../Polynomial_Calc/SiLU.h"
#include <iostream>
#include <chrono>
#include <ctime>
#define BATCH_SIZE 40
using namespace troy;
using namespace std;

/*
    Try to calculate the cipher with a fixed polynomial 
*/ 
void approx_with_fix(const CKKSEncoder &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const double &scale,
                    const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {
    
    // Prepare the plaintext for 2, 5, and 1
    Plaintext plain_2, plain_5, plain_1;
    encoder.encode_float64_single(2.0, std::nullopt, scale, plain_2);
    encoder.encode_float64_single(5.0, std::nullopt, scale, plain_5);
    encoder.encode_float64_single(1.0, std::nullopt, scale, plain_1);

    // Compute x + 2
    Ciphertext x_plus_2 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_2, plain_2);

    // Compute x + 5
    Ciphertext x_plus_5 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_5, plain_5);

    // Compute x + 1
    Ciphertext x_plus_1 = x_encrypted;
    evaluator.add_plain_inplace(x_plus_1, plain_1);

    // Compute (x + 2) * (x + 5)
    evaluator.multiply_inplace(x_plus_2, x_plus_5);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Compute (x + 2) * (x + 5) * (x + 1)
    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Compute (x + 2) * (x + 5) * (x + 1) * (x + 1)
    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    evaluator.mod_switch_to_inplace(x_plus_1, x_plus_2.parms_id());
    x_plus_1.scale() = x_plus_2.scale();
    evaluator.multiply_inplace(x_plus_2, x_plus_1);
    evaluator.relinearize_inplace(x_plus_2, relin_keys);
    evaluator.rescale_to_next_inplace(x_plus_2);

    // Store the result
    encrypted_result = x_plus_2;
}


void horner(const CKKSEncoder &encoder, const Evaluator &evaluator, const RelinKeys &relin_keys, const double &scale,
            const Polynomial<double> poly, const Ciphertext &x_encrypted, Ciphertext &encrypted_result) {

    /*
        Conduct the horner's algorithm in linear complexity
    */
    Ciphertext my_cipher = x_encrypted; // set the initial value to be x
    size_t poly_deg = poly.get_degree();
    Plaintext plain_coeff;
    double largest_coeff = poly.get_coeff_by_rank(poly_deg);
    //double largest_coeff = 1.0;
    encoder.encode_float64_single(largest_coeff, std::nullopt, scale, plain_coeff);
    evaluator.multiply_plain_inplace(my_cipher, plain_coeff);
    evaluator.rescale_to_next_inplace(my_cipher); // a_n*x

    for (int i = poly_deg - 1; i >= 0; i--) {
        if (i != poly_deg - 1) {
            // multiply x
            Ciphertext this_x = x_encrypted;
            // First change scale 
            evaluator.mod_switch_to_inplace(this_x, my_cipher.parms_id());
            this_x.scale() = my_cipher.scale();
            // Second multiply
            evaluator.multiply_inplace(my_cipher, this_x);
            evaluator.relinearize_inplace(my_cipher, relin_keys);
            evaluator.rescale_to_next_inplace(my_cipher);
        }
        // Add const
        Plaintext this_coeff;
        encoder.encode_float64_single(poly.get_coeff_by_rank((size_t)(i)), std::nullopt, my_cipher.scale(), this_coeff);
        //encoder.encode_float64_single(0.0, std::nullopt, my_cipher.scale(), this_coeff);
        evaluator.add_plain_inplace(my_cipher, this_coeff);
    }

    encrypted_result = my_cipher;
}


int main() {

    int deg;
    cout << "Input deg: " << endl;
    cin >> deg;

    vector<size_t> mod_chain;
    for (int i = 0; i < deg + 2; i++) {
        if (i == 0) {
            mod_chain.push_back(60);
        }
        else if (i == deg + 1) {
            mod_chain.push_back(60);
        }
        else {
            mod_chain.push_back(40);
        }
    }

    EncryptionParameters parms(SchemeType::CKKS);

    Remez<double, double> my_p(deg, gelu_and_sqplus);
    Polynomial<double> poly = my_p.generate_approx(deg, 0);
    poly.prune();
    poly.check();

    size_t poly_modulus_degree = 32768;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::create(poly_modulus_degree, mod_chain));
    // Change this modulus chain if we need higher degree


    double scale = pow(2.0, 40);

    auto context = HeContext::create(parms, true, SecurityLevel::Classical128);
    print_parameters(*context);
    cout << endl;

    CKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();
    //cout << "Number of slots: " << slot_count << endl;

    context->to_device_inplace();
    encoder.to_device_inplace();

    KeyGenerator keygen(context);
    auto secret_key = keygen.secret_key();
    PublicKey public_key = keygen.create_public_key(false);
    RelinKeys relin_keys = keygen.create_relin_keys(false);
    GaloisKeys gal_keys = keygen.create_galois_keys(false);
    Encryptor encryptor(context); encryptor.set_public_key(public_key);
    Evaluator evaluator(context);
    Decryptor decryptor(context, secret_key);


    vector<complex<double>> input;
    input.reserve(slot_count);
    double curr_point = -5;
    //double step_size = 1.0 / (static_cast<double>(slot_count) - 1);
    double step_size = (double)10 / (double)BATCH_SIZE;
    for (size_t i = 0; i < BATCH_SIZE; i++)
    {
        input.push_back(curr_point);
        curr_point += step_size;
    }
    cout << "Input vector: " << endl;
    print_vector(input, BATCH_SIZE, 7);

    /*
        Try to calculate th
    */


    Plaintext x_plain;
    cout << "Encode input vectors." << endl;
    encoder.encode_complex64_simd(input, std::nullopt, scale, x_plain);
    Ciphertext x_encrypted;
    encryptor.encrypt_asymmetric(x_plain, x_encrypted);
    Ciphertext encrypted_result;
    //approx_with_fix(encoder, evaluator, relin_keys, scale, x_encrypted, encrypted_result);
    auto start = std::chrono::high_resolution_clock::now();
    horner(encoder, evaluator, relin_keys, scale, poly, x_encrypted, encrypted_result);
    
    /*
    Decrypt, decode, and print the result.
    */
    Plaintext plain_result;
    decryptor.decrypt(encrypted_result, plain_result);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    vector<complex<double>> result;
    encoder.decode_complex64_simd(plain_result, result);
    //print_vector(result, 16, 7);
    for (int i = 0; i < BATCH_SIZE; i++) {
        //cout << "x: " << input[i].real() << "       Relu(x): " << relu(input[i].real()) << "      SiLU(x): " << silu(input[i].real()) << "       计算结果：" << result[i].real() << endl;
        //cout << "x: " << input[i].real() << "       Relu(x): " << relu(input[i].real()) << "    abs(x): " << abs_test(input[i].real()) << "       计算结果：" << result[i].real() << endl;
        cout << "x: " << input[i].real() << "       Relu(x): " << relu(input[i].real()) << "       计算结果：" << result[i].real() << endl;
    }
    std::cout << "运行时间: " << (double)duration.count() / (double)1000 << " ms" << std::endl;
    cout << endl;
    return 0;

}
